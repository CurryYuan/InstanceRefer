
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

//hashing
//input N*F float tensor, pointer to output N'*F int64 tensor, N*1 count tensor, N*1 index tensor
__global__ void convert_map_kernel(int k, int N, int N_nonzero, const int *__restrict__ nmap, const int *__restrict__ idx_batch, const int *__restrict__ idx_point, int *__restrict__ out){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(index < N_nonzero){
        int i = idx_batch[index];
        int j = idx_point[index];
        out[index << 1] = nmap[i * N + j];
        out[(index << 1) + 1] = j;
    }
}



void convert_map_wrapper(int k, int N, int N_nonzero, const int * nmap, const int * idx_batch, const int * idx_point, int * out){
    convert_map_kernel<<<int(ceil(N_nonzero / 512))+1, 512>>>(k, N, N_nonzero, nmap, idx_batch, idx_point, out);
}
