
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

//counting
//input N*3 int32 tensor output N*1 int64 tensor
__global__ void count_kernel(int N, const int *__restrict__ data, int *__restrict__ out){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N){
        if(data[i] >= 0) atomicAdd(&out[data[i]], 1);
    }
}


void count_wrapper(int N, const int * data, int * out){
    count_kernel<<<ceil((double)N/512), 512>>>(N, data, out);
}
